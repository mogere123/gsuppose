#include "hip/hip_runtime.h"
#include "catmu.h"
#include <omp.h>

// Kernel de CUDA para la convolución TMU 2D
__global__ void lutKernel2D(sImage image, sPositions pos, sLutPSF psf,
                            hipTextureObject_t texPSF,
                            int offset_image, int offset_position){

    // Identificación del kernel
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    float px, py, pixel;
    float factor_x, factor_y, center_x, center_y;

    // Factor de conversión entre el pixel de la PSF y el de la imagen
    factor_x = image.pixel_width / psf.pixel_width;
    factor_y = image.pixel_height / psf.pixel_height;

    // Centro de la PSF
    center_x = psf.width / 2.0;
    center_y = psf.height / 2.0;

    // Condición para calcular el pixel (que pertenezca a la imagen)
    if (idx < image.width && idy < image.height) {

        // Resultado acumulado (inicialmente en cero)
        pixel = 0;

        // Iteración sobre todas las fuentes virtuales
        for (int i = 0; i < pos.n; i++){
            // Conversión de coordenadas
            px = (idx-pos.data[offset_position + i*2]) * factor_x + center_x;
            py = (idy-pos.data[offset_position + i*2+1]) * factor_y + center_y;
            // Evaluación realizada por la TMU para las coordenadas dadas
            pixel += tex2D<float>(texPSF, px, py);
        }

        // Resultado aplicado a la imagen
        image.data[offset_image + idy * image.width + idx] = pixel;
    }

}

// Kernel de CUDA para la convolución TMU 3D
__global__ void lutKernel3D(sImage image, sPositions pos, sLutPSF psf,
                            hipTextureObject_t texPSF,
                            int offset_image, int offset_position){

    // Identificación del kernel
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;
    size_t idz = blockIdx.z * blockDim.z + threadIdx.z;

    float px, py, pz, pixel;
    float factor_x, factor_y, factor_z, center_x, center_y, center_z;

    // Factor de conversión entre el pixel de la PSF y el de la imagen
    factor_x = image.pixel_width / psf.pixel_width;
    factor_y = image.pixel_height / psf.pixel_height;
    factor_z = image.pixel_depth / psf.pixel_depth;

    // Centro de la PSF
    center_x = psf.width / 2.0;
    center_y = psf.height / 2.0;
    center_z = psf.depth / 2.0;

    // Condición para calcular el pixel (que pertenezca a la imagen)
    if (idx < image.width && idy < image.height && idz < image.depth) {

        // Resultado acumulado (inicialmente en cero)
        pixel = 0;

        // Iteración sobre todas las fuentes virtuales
        for (int i = 0; i < pos.n; i++){
            // Conversión de coordenadas
            px = (idx-pos.data[offset_position + i*3]) * factor_x + center_x;
            py = (idy-pos.data[offset_position + i*3+1]) * factor_y + center_y;
            pz = (idz-pos.data[offset_position + i*3+2]) * factor_z + center_z;
            // Evaluación realizada por la TMU para las coordenadas dadas
            pixel += tex3D<float>(texPSF, px, py, pz);
        }

        // Resultado aplicado a la imagen
        image.data[(offset_image +
                    idz * image.height * image.width +
                    idy * image.width + idx)] = pixel;
    }

}

// Configuración del dispositivo (GPU) utilizado
int set_device(int device){
    int count, current_device;
    // Consulta la cantidad de dispositivos disponibles
    hipGetDeviceCount(&count);

    // Revisa que el dispositivo seleccionado exista
    if (device >= count){
        return 101;
    }
    info_print("Selecting device %d (%d available)\n", device, count);

    // Configura el dispositivo
    hipSetDevice(device);
    CUDA_CHECK_ERROR(return err);

    // Consulta el dispositivo actual
    hipGetDevice(&current_device);
    info_print("Current device: %d\n", current_device);

    // Reporta el error en caso de que la asignación falle
    if (current_device != device){
        return -2;
    }

    return 0;
}

extern "C" {
int get_available_devices(int * count){
    hipGetDeviceCount(count);
    CUDA_CHECK_ERROR(return err);
    return 0;
}}

int printDevProp(hipDeviceProp_t devProp){
    printf("  Name:                                  %s\n",  devProp.name);
    printf("  Number of multiprocessors:             %d\n",  devProp.multiProcessorCount);
    printf("  Total global memory (Bytes):           %lu\n",  devProp.totalGlobalMem);
    printf("  Total shared memory per block (Bytes): %lu\n",  devProp.sharedMemPerBlock);
    printf("  Major revision number:                 %d\n",  devProp.major);
    printf("  Minor revision number:                 %d\n",  devProp.minor);
    printf("  Total registers per block:             %d\n",  devProp.regsPerBlock);
    printf("  Warp size:                             %d\n",  devProp.warpSize);
    printf("  Maximum memory pitch:                  %lu\n",  devProp.memPitch);
    printf("  Maximum threads per block:             %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("  Maximum dimension %d of block:          %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("  Maximum dimension %d of grid:           %d\n", i, devProp.maxGridSize[i]);
    printf("  Clock rate (KHz):                      %d\n",  devProp.clockRate);
    printf("  Total constant memory (Bytes):         %lu\n",  devProp.totalConstMem);
    printf("  Texture alignment:                     %lu\n",  devProp.textureAlignment);
    printf("  Concurrent copy and execution:         %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("  Kernel execution timeout:              %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return 0;
}

extern "C" {
int getDevProp(int device, cudaDevicePropCatmu * devProp){
    hipDeviceProp_t devPropCuda;
    hipGetDeviceProperties(&devPropCuda, device);

    strcpy(devProp->name, devPropCuda.name);
    devProp->multiProcessorCount  = devPropCuda.multiProcessorCount;
    devProp->totalGlobalMem  = devPropCuda.totalGlobalMem;
    devProp->sharedMemPerBlock  = devPropCuda.sharedMemPerBlock;
    devProp->major  = devPropCuda.major;
    devProp->minor  = devPropCuda.minor;
    devProp->regsPerBlock  = devPropCuda.regsPerBlock;
    devProp->warpSize  = devPropCuda.warpSize;
    devProp->memPitch  = devPropCuda.memPitch;
    devProp->maxThreadsPerBlock  = devPropCuda.maxThreadsPerBlock;
    devProp->maxThreadsDim[0]  = devPropCuda.maxThreadsDim[0];
    devProp->maxThreadsDim[1]  = devPropCuda.maxThreadsDim[1];
    devProp->maxThreadsDim[2]  = devPropCuda.maxThreadsDim[2];
    devProp->maxGridSize[0]  = devPropCuda.maxGridSize[0];
    devProp->maxGridSize[1]  = devPropCuda.maxGridSize[1];
    devProp->maxGridSize[2]  = devPropCuda.maxGridSize[2];
    devProp->clockRate  = devPropCuda.clockRate;
    devProp->totalConstMem  = devPropCuda.totalConstMem;
    devProp->textureAlignment  = devPropCuda.textureAlignment;
    devProp->deviceOverlap  = devPropCuda.deviceOverlap;
    devProp->kernelExecTimeoutEnabled  = devPropCuda.kernelExecTimeoutEnabled;

    return 0;
}}

extern "C" {
int query_devices(){
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("\nCUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }
    return 0;
}}


int free_device_memory(sImage * d_image, sPositions * d_pos,
                       hipTextureObject_t * texObj, hipArray * cuArray){

    if (d_image->data != 0){
        info_print("Free results memory\n");
        hipFree(d_image->data);
        CUDA_CHECK_ERROR();
    }

    if (d_pos->data != 0){
        info_print("Free positions memory\n");
        hipFree(d_pos->data);
        CUDA_CHECK_ERROR();
    }

    info_print("Destroy texture object\n");
    hipDestroyTextureObject(*texObj);
    CUDA_CHECK_ERROR();
    info_print("Free texture memory\n");
    hipFreeArray(cuArray);
    CUDA_CHECK_ERROR();

    return 0;
}

int set_texture_2d(hipArray * cuArray, sLutPSF * psf,
                  hipTextureObject_t * texObj){

    // Formato del array que contiene la textura (formalismo, acá no hay canales RGBA)
    info_print("Allocate CUDA array in device memory\n");
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0,
                                     hipChannelFormatKindFloat);

    // Pedido de memoria para el array de la textura
    hipMallocArray(&cuArray, &channelDesc, psf->width, psf->height);
    CUDA_CHECK_ERROR(return err);

    // Transferencia de CPU a GPU de la PSF en formato array (compatible para texturas)
    info_print("Copy to device memory sLutPSF data from host\n");
    hipMemcpy2DToArray(cuArray, 0, 0, psf->data,
                        psf->width * sizeof(float),
                        psf->width * sizeof(float),
                        psf->height,
                        hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(return err);

    // Configuración del descriptor de la textura
    info_print("Specify texture\n");
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;
    CUDA_CHECK_ERROR(return err);

    // Parámetros configurables
    info_print("Specify texture object parameters\n");
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    // Modo de acceso (border -> cualquier acceso fuera de rango devuelve un cero)
    texDesc.addressMode[0]   = hipAddressModeBorder;
    texDesc.addressMode[1]   = hipAddressModeBorder;
    // Filtrado (interpolado lineal de valores)
    texDesc.filterMode       = hipFilterModeLinear;
    // Tipo de acceso (basado en los elementos del array -> float32)
    texDesc.readMode         = hipReadModeElementType;
    // Tipo de coordenadas no normalizadas -> [0, N)
    texDesc.normalizedCoords = 0;
    CUDA_CHECK_ERROR(return err);

    // Objecto que representa la textura -> texObj
    info_print("Create texture object\n");
    hipCreateTextureObject(texObj, &resDesc, &texDesc, NULL);
    CUDA_CHECK_ERROR(return err);

    return 0;

}

int set_texture_3d(hipArray * cuArray, sLutPSF * psf,
                  hipTextureObject_t * texObj){

    // Formato del array que contiene la textura (formalismo, acá no hay canales RGBA)
    info_print("Allocate CUDA array in device memory\n");
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0,
                                     hipChannelFormatKindFloat);

    hipExtent extent;
    extent.depth = psf->depth;
    extent.height = psf->height;
    extent.width = psf->width;

    // Pedido de memoria para el array de la textura
    hipMalloc3DArray(&cuArray, &channelDesc, extent);
    CUDA_CHECK_ERROR(return err);

    // Transferencia de CPU a GPU de la PSF en formato array (compatible para texturas)
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(psf->data,
                                              extent.width*sizeof(float),
                                              extent.width,
                                              extent.height);
    copyParams.dstArray = cuArray;
    copyParams.extent   = extent;
    copyParams.kind     = hipMemcpyHostToDevice;
    info_print("Copy to device memory sLutPSF3d data from host\n");
    hipMemcpy3D(&copyParams);
    CUDA_CHECK_ERROR(return err);

    // Configuración del descriptor de la textura
    info_print("Specify texture\n");
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;
    CUDA_CHECK_ERROR(return err);

    // Parámetros configurables
    info_print("Specify texture object parameters\n");
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    // Modo de acceso (border -> cualquier acceso fuera de rango devuelve un cero)
    texDesc.addressMode[0]   = hipAddressModeBorder;
    texDesc.addressMode[1]   = hipAddressModeBorder;
    texDesc.addressMode[2]   = hipAddressModeBorder;
    // Filtrado (interpolado lineal de valores)
    texDesc.filterMode       = hipFilterModeLinear;
    // Tipo de acceso (basado en los elementos del array -> float32)
    texDesc.readMode         = hipReadModeElementType;
    // Tipo de coordenadas no normalizadas -> [0, N)
    texDesc.normalizedCoords = 0;
    CUDA_CHECK_ERROR(return err);

    // Objecto que representa la textura -> texObj
    info_print("Create texture object\n");
    hipCreateTextureObject(texObj, &resDesc, &texDesc, NULL);
    CUDA_CHECK_ERROR(return err);

    return 0;

}

int set_images(sImage * h_image, sImage * d_image){
    info_print("Allocating memory for results in device\n");

    // Calcula el tamaño requerido
    size_t size = h_image->width * h_image->height * h_image->depth * h_image->N * sizeof(float);
    info_print("Request: %lu bytes\n", size);

    // Revisa si la memoria fue asignada antes
    if (d_image->data != 0){
        info_print("Memory allocated in device: %lu bytes\n", d_image->allocated_size);

        // Revisa que los tamaños coincidan
        if (d_image->allocated_size != size){
            // Libera el bloque de tamaño incorrecto
            info_print("Releasing block\n");
            hipFree(d_image->data);
            CUDA_CHECK_ERROR(return err);
            info_print("Memory released\n");

            // Pide el bloque correcto
            info_print("Allocating memory block of %lu bytes\n", size);
            hipMalloc(&d_image->data, size);
            CUDA_CHECK_ERROR(return err);
            d_image->allocated_size = size;
            info_print("Memory allocated at %p\n", d_image->data);
        }

    } else {
        // Pide memoria
        info_print("Allocating memory block of %lu bytes\n", size);
        hipMalloc(&d_image->data, size);
        CUDA_CHECK_ERROR(return err);
        d_image->allocated_size = size;
        info_print("Memory allocated at %p\n", d_image->data);
    }

    // Actualiza la metadata de las imágenes
    info_print("Updating image metadata\n");
    d_image->width = h_image->width;
    d_image->height = h_image->height;
    d_image->depth = h_image->depth;
    d_image->pixel_width = h_image->pixel_width;
    d_image->pixel_height = h_image->pixel_height;
    d_image->pixel_depth = h_image->pixel_depth;
    info_print("width = %d\n", d_image->width);
    info_print("height = %d\n", d_image->height);
    info_print("depth = %d\n", d_image->depth);
    info_print("pixel_width = %f\n", d_image->pixel_width);
    info_print("pixel_height = %f\n", d_image->pixel_height);
    info_print("pixel_depth = %f\n", d_image->pixel_depth);

    info_print("Memory for results is ready to use\n");
    return 0;
}

int upload_positions(sPositions * h_pos, sPositions * d_pos){
    info_print("Allocating memory for lists of positions in device\n");

    // Calcula el tamaño requerido
    size_t size = h_pos->N * h_pos->n * h_pos->dim * sizeof(float);
    info_print("Request: %lu bytes\n", size);

    // Revisa si la memoria fue asignada antes
    if (d_pos->data != 0){
        info_print("Memory allocated in device: %lu bytes\n", d_pos->allocated_size);

        // Revisa si la memoria fue asignada antes
        if (d_pos->allocated_size != size){
            // Libera el bloque de tamaño incorrecto
            info_print("Releasing block\n");
            hipFree(d_pos->data);
            CUDA_CHECK_ERROR(return err);
            info_print("Memory released\n");

            // Pide el bloque correcto
            info_print("Allocating memory block of %lu bytes\n", size);
            hipMalloc(&d_pos->data, size);
            CUDA_CHECK_ERROR(return err);
            d_pos->allocated_size = size;
            info_print("Memory allocated at %p\n", d_pos->data);
        }
    } else {
        // Pide memoria
        info_print("Allocating memory block of %lu bytes\n", size);
        hipMalloc(&d_pos->data, size);
        CUDA_CHECK_ERROR(return err);
        d_pos->allocated_size = size;
        info_print("Memory allocated at %p\n", d_pos->data);
    }

    d_pos->n = h_pos->n;
    info_print("n = %d\n", d_pos->n);
    info_print("Memory for positions is ready to use\n");

    // Tranfiere los datos de CPU a GPU
    info_print("Uploading positions from host CPU to device GPU\n");
    hipMemcpy(d_pos->data, h_pos->data, size, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(return err);
    info_print("Successfully uploaded\n");

    return 0;
}

int download_results(sImage * h_image, sImage * d_image){
    // Descarga los resultados de GPU a CPU
    info_print("Downloading results from device GPU to host CPU\n");
    hipMemcpy(h_image->data, d_image->data, d_image->allocated_size, hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR(return err);
    info_print("Successfully downloaded\n");

    return 0;
}

extern "C" {
int lutConvolution(callback checkpoint, sImage * h_image, sPositions * h_pos, sLutPSF * h_psf, sConfig * config){
    info_print("Starting LUT Convolution Module of CaTMU\n");

    // Definición de variables
    int r = 0;
    clock_t start, end;
    double elapsed_time = 0;
    unsigned long loop_counter = 0;

    hipArray * cuArray = 0;
    hipTextureObject_t texObj = 0;

    hipStream_t * stream;

    sImage d_image;
    sPositions d_pos;

    d_image.data = 0;
    d_pos.data = 0;

    // Configuración del dispositivo
    info_print("Setting device\n");
    r = set_device(config->device);
    if (r != 0) return r;

    // Configuración de la textura
    info_print("Setting PSF as texture object\n");
    if (h_psf->dim == 2) {
        info_print("PSF TYPE: LUT 2D\n");
        r = set_texture_2d(cuArray, h_psf, &texObj);
    }
    if (h_psf->dim == 3) {
        info_print("PSF TYPE: LUT 3D\n");
        r = set_texture_3d(cuArray, h_psf, &texObj);
    }

    if (r != 0) return r;

    info_print("width = %d\n", h_psf->width);
    info_print("height = %d\n", h_psf->height);
    info_print("depth = %d\n", h_psf->depth);
    info_print("pixel_width = %f\n", h_psf->pixel_width);
    info_print("pixel_height = %f\n", h_psf->pixel_height);
    info_print("pixel_depth = %f\n", h_psf->pixel_depth);

    // Bucle principal para mantener la GPU en espera entre pedidos
    info_print("Main loop\n");
    while (checkpoint(elapsed_time, loop_counter) == true){

        // Control de tiempos (para rendimiento)
        start = clock();

        // Configuración de la memoria para resultados
        info_print("Setting memory for results\n");
        r = set_images(h_image, &d_image);
        if (r != 0) return r;

        // Configuración de la memoria para posiciones y subida de datos a la GPU
        info_print("Uploading data for positions\n");
        info_print("dim -> %d\n", h_pos->dim);
        info_print("n -> %d\n", h_pos->n);
        r = upload_positions(h_pos, &d_pos);
        if (r != 0) return r;

        if (h_psf->dim != h_pos->dim) {
            error_print("Incompatible dimensions: psf%dD vs Position%dD\n",
                        h_psf->dim, h_pos->dim);
            return -1;
        }

        // Configuración de los streams de CUDA (para optimizar el uso de la GPU)
        stream = (hipStream_t *) malloc(config->n_streams * sizeof(hipStream_t));
        info_print("Setting %d streams\n", config->n_streams);
        for (int i = 0; i < config->n_streams; i ++)
        {
            hipStreamCreate(&stream[i]);
            CUDA_CHECK_ERROR(return err);
            info_print("Stream %d created\n", i);
        }

        // Configuración de los bloques y grillas para la paralelización
        info_print("Grid and block sizes:\n");

        dim3 dimBlock;
        dim3 dimGrid;

        if (h_pos->dim == 2) {
            dimBlock.x = config->block_size;
            dimBlock.y = config->block_size;
            dimGrid.x = (h_image->width + dimBlock.x - 1) / dimBlock.x;
            dimGrid.y = (h_image->height + dimBlock.y - 1) / dimBlock.y;

            info_print("dimGrid: %dx%d\n", dimGrid.x, dimGrid.y);
            info_print("dimBlock: %dx%d\n", dimBlock.x, dimBlock.y);
        }
        if (h_pos->dim == 3) {
            dimBlock.x = config->block_size;
            dimBlock.y = config->block_size;
            dimBlock.z = 1;

            dimGrid.x = (h_image->width + dimBlock.x - 1) / dimBlock.x;
            dimGrid.y = (h_image->height + dimBlock.y - 1) / dimBlock.y;
            dimGrid.z = (h_image->depth + dimBlock.z - 1) / dimBlock.z;

            info_print("dimGrid: %dx%dx%d\n", dimGrid.x, dimGrid.y, dimGrid.z);
            info_print("dimBlock: %dx%dx%d\n", dimBlock.x, dimBlock.y, dimBlock.z);
        }

        // Offsets aplicados a los punteros de imagenes y posiciones para acceder a cada uno de ellos
        int offset_image = 0;
        int offset_position = 0;

        // Carga de tareas para cada stream (iterando entre streams)
        info_print("Launching streams\n");
        for (int i = 0; i < h_pos->N; i++)
        {
            // Instancia de kernels en diferentes streams
            info_print("Convolution %d launched on stream %d\n", i, i % config->n_streams);
            info_print("Image offset: %d, Position offset: %d\n", offset_image, offset_position);
            if (h_pos->dim == 2) {
                lutKernel2D <<<dimGrid, dimBlock, 0, stream[i % config->n_streams]>>> \
                (d_image, d_pos, *h_psf, texObj, offset_image, offset_position);
            }
            if (h_pos->dim == 3) {
                lutKernel3D <<<dimGrid, dimBlock, 0, stream[i % config->n_streams]>>> \
                (d_image, d_pos, *h_psf, texObj, offset_image, offset_position);
            }
            CUDA_CHECK_ERROR(return err);

            // Actualización del buffer para la siguiente convolución
            offset_position += h_pos->dim * h_pos->n;
            offset_image += h_image->width * h_image->height * h_image->depth;
        }

        // Barrera de sincronización (todos los streams deben terminar acá)
        info_print("Waiting for synchronization...\n");
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR(return err);
        info_print("Synchronization completed\n");

        // Descarga de resultados
        download_results(h_image, &d_image);

        // Cierre de streams generados
        info_print("Destroying %d streams\n", config->n_streams);
        for (int i = 0; i < config->n_streams; i ++)
        {
            hipStreamDestroy(stream[i]);
            CUDA_CHECK_ERROR(return err);
            info_print("Stream %d destroyed\n", i);
        }
        free(stream);

        // Marca final de tiempo, actualización del tiempo transcurrido e incremento del contador
        end = clock();
        elapsed_time = ((double) (end - start)) / CLOCKS_PER_SEC;
        loop_counter += 1;
    }

    // Memoria liberada en el dispositivo (general)
    info_print("Releasing memory from device\n");
    free_device_memory(&d_image, &d_pos, &texObj, cuArray);

    // Reseteo del entorno creado (para mayor seguridad, no debería ser necesario)
    info_print("Executing reset of device\n");
    hipDeviceReset();
    CUDA_CHECK_ERROR(return err);
    info_print("\nShutting down\n");

    return 0;
}}

double get_T(sLutPSF * psf, int i, int j){
    if (j >= 0 && j < psf->width && i >= 0 && i < psf->height){
        return psf->data[i * psf->width + j];
    } else {
        return 0;
    }

}

double cpu_tex2d(sLutPSF * psf, float x, float y){
    int i, j;
    double a, b;
    double T1, T2, T3, T4;

    i = (int) floor(y - 0.5);
    a = (double) (y - 0.5) - i;

    j = (int) floor(x - 0.5);
    b = (double) (x - 0.5) - j;

    T1 = get_T(psf, i, j);
    T2 = get_T(psf, i+1, j);
    T3 = get_T(psf, i, j+1);
    T4 = get_T(psf, i+1, j+1);

    return (double) ((1-a) * (1-b) * T1 + a * (1-b) * T2 + (1-a) * b * T3 + a * b * T4);

}

extern "C" {
int cpu_lut_convolve2D_openmp(sImage * image, sPositions * positions, sLutPSF * psf){
    info_print("Starting LUT Convolution over CPU powered by Open MP\n");
    float px, py, pixel;
    float factor_x, factor_y, center_x, center_y;
    int offset_position = positions->n * 2;
    int offset_image = image->width * image->height;

    // Factor de conversión entre el pixel de la PSF y el de la imagen
    factor_x = image->pixel_width / psf->pixel_width;
    factor_y = image->pixel_height / psf->pixel_height;

    // Centro de la PSF
    center_x = psf->width / 2.0;
    center_y = psf->height / 2.0;

    #pragma omp parallel for private(pixel) private(px) private(py)
    for (int k=0; k<positions->N; k++){
        info_print("Thread %d assigned to convolution %d of %d\n", omp_get_thread_num(), k, positions->N);
        for (int i=0; i<image->width; i++){
            for (int j=0; j<image->height; j++){
                pixel = 0;
                for (int l=0; l<positions->n; l++){
                    px = (i-positions->data[k * offset_position + l*2]) * factor_x + center_x;
                    py = (j-positions->data[k * offset_position + l*2+1]) * factor_y + center_y;

                    // Evaluación realizada por la TMU simulada en CPU para las coordenadas dadas
                    pixel += cpu_tex2d(psf, px, py);
                }
                image->data[k * offset_image + j * image->width + i] = pixel;
            }
        }
    }

    return 0;
}}

extern "C" {
int cpu_lut_convolve2D(sImage * image, sPositions * positions, sLutPSF * psf){
    info_print("Starting LUT Convolution over CPU without Open MP\n");
    float px, py, pixel;
    float factor_x, factor_y, center_x, center_y;
    int offset_position = positions->n * 2;
    int offset_image = image->width * image->height;

    // Factor de conversión entre el pixel de la PSF y el de la imagen
    factor_x = image->pixel_width / psf->pixel_width;
    factor_y = image->pixel_height / psf->pixel_height;

    // Centro de la PSF
    center_x = psf->width / 2.0;
    center_y = psf->height / 2.0;

    for (int k=0; k<positions->N; k++){
        info_print("Convolution %d of %d in progress\n", k, positions->N);
        for (int i=0; i<image->width; i++){
            for (int j=0; j<image->height; j++){
                pixel = 0;
                for (int l=0; l<positions->n; l++){
                    px = (i-positions->data[k * offset_position + l*2]) * factor_x + center_x;
                    py = (j-positions->data[k * offset_position + l*2+1]) * factor_y + center_y;

                    // Evaluación realizada por la TMU simulada en CPU para las coordenadas dadas
                    pixel += cpu_tex2d(psf, px, py);
                }
                image->data[k * offset_image + j * image->width + i] = pixel;
            }
        }
    }

    return 0;
}}

float gaussian_2d(float x, float y, float * params){
    return params[0] * exp(-(x*x + y*y) / params[1] / 2);
}

int evaluate_psf_2d(sLutPSF * psf, psf_function * f, float * params){
    float x, y;
    for (float i=0; i<psf->width; i++){
        for (float j=0; j<psf->height; j++){
            x = i - (psf->width / 2.0);
            y = j - (psf->height / 2.0);
            psf->data[(int) j * psf->width + (int) i] = (*f)(x, y, params);
        }
    }
    return 0;
}
